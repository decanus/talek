
#include <hip/hip_runtime.h>
/***************************************************
* Module for PIR
*
* To be compiled with nvcc -ptx pir.cu
* Debug: nvcc -arch=sm_20 -ptx pir.cu
* Note: CUDA may not support all versions of gcc;
* See
* https://groups.google.com/forum/#!topic/torch7/WaNmWZqMnzw
**************************************************/

//#include <stdio.h>

#ifdef __cplusplus
extern "C" {
#endif

typedef char int8_cu;
typedef unsigned char uint8_cu;
typedef long int int32_cu;
typedef unsigned long int uint32_cu;
typedef long long int int64_cu;
typedef unsigned long long int uint64_cu;
#define DATA_TYPE uint64_cu

// CUDA Kernel
__global__
void pir(DATA_TYPE* db,
        uint8_cu* reqs,
        DATA_TYPE* output,
        //DATA_TYPE* scratch,
        uint32_cu batchSize,
        uint32_cu reqLength,
        uint32_cu numBuckets,
        uint32_cu bucketSize,
        uint32_cu globalSize){
        //uint32_cu scratchSize) {
  //int localIndex = threadIdx.x;
  //int groupIndex = blockIdx.x;
  uint32_cu globalIndex = threadIdx.x + (blockIdx.x * blockDim.x);

  if (globalIndex >= globalSize) {
    return;
  }
  // Iterate over all buckets, xor data into my result
  DATA_TYPE result = 0;
  uint32_cu reqIndex = (globalIndex / bucketSize) * reqLength;
  uint32_cu offset = globalIndex % bucketSize;
  uint8_cu reqBit;
  for (uint32_cu i = 0; i < numBuckets; i++) {
    reqBit = reqs[reqIndex + (i/8)] & (1 << (i%8));
    if (reqBit > 0) {
      result ^= db[i*bucketSize+offset];
    }
  }
  output[globalIndex] = result;
}

#ifdef __cplusplus
}
#endif
