
#include <hip/hip_runtime.h>
/***************************************************
* Module for PIR
*
* To be compiled with nvcc -ptx pir.cu
* Debug: nvcc -arch=sm_20 -ptx pir.cu
* Note: CUDA may not support all versions of gcc;
* See
* https://groups.google.com/forum/#!topic/torch7/WaNmWZqMnzw
**************************************************/

//#include <stdio.h>

#ifdef __cplusplus
extern "C" {
#endif

typedef long int int32_cu;
typedef unsigned long int uint32_cu;
typedef long long int int64_cu;
typedef unsigned long long int uint64_cu;
#define DATA_TYPE uint64_cu

// CUDA Kernel
__global__
void pir(DATA_TYPE* db,
        char* reqs,
        DATA_TYPE* output,
        //DATA_TYPE* scratch,
        uint32_cu batchSize,
        uint32_cu reqLength,
        uint32_cu numBuckets,
        uint32_cu bucketSize,
        uint32_cu globalSize,
        uint32_cu scratchSize) {
  //int localIndex = threadIdx.x;
  //int groupIndex = blockIdx.x;
  int globalIndex = threadIdx.x + (blockIdx.x * blockDim.x);

  if (globalIndex >= globalSize) {
    return;
  }
  // Iterate over all buckets, xor data into my result
  DATA_TYPE result = 0;
  int reqIndex = (globalIndex / bucketSize) * reqLength;
  int offset = globalIndex % bucketSize;
  unsigned char reqBit;
  for (int i = 0; i < numBuckets; i++) {
    reqBit = reqs[reqIndex + (i/8)] & (1 << (i%8));
    if (reqBit > 0) {
      result ^= db[i*bucketSize+offset];
    }
  }
  output[globalIndex] = result;
}

#ifdef __cplusplus
}
#endif
