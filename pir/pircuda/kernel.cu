
#include <hip/hip_runtime.h>
/***************************************************
* Module for PIR
*
* To be compiled with nvcc -ptx pir.cu
* Debug: nvcc -arch=sm_20 -ptx pir.cu
* Note: CUDA may not support all versions of gcc;
* See
* https://groups.google.com/forum/#!topic/torch7/WaNmWZqMnzw
**************************************************/

//#include <stdio.h>

#ifdef __cplusplus
extern "C" {
#endif

typedef char int8_cu;
typedef unsigned char uint8_cu;
typedef long int int32_cu;
typedef unsigned long int uint32_cu;
typedef long long int int64_cu;
typedef unsigned long long int uint64_cu;
#define DATA_TYPE uint64_cu

// CUDA Kernel
__global__
void pir(DATA_TYPE* db,
        uint8_cu* reqs,
        DATA_TYPE* output,
        int batchSize,
        int reqLength,
        int numBuckets,
        int bucketSize,
        int globalSize){
  //int localIndex = threadIdx.x;
  //int groupIndex = blockIdx.x;
  int globalIndex = threadIdx.x + (blockIdx.x * blockDim.x);

  if (globalIndex >= globalSize) {
    return;
  }
  __syncthreads();

  // Iterate over requests in a batch, atomic_xor my data into output
  int bucketId = globalIndex / bucketSize;
  int depthOffset = globalIndex % bucketSize;
  DATA_TYPE data = db[globalIndex];
  DATA_TYPE* addr;
  uint8_cu reqBit;
  for (int i = 0; i < batchSize; i++) {
    reqBit = reqs[(i*reqLength) + (bucketId/8)] & (1 << (bucketId%8));
    if (reqBit > 0) {
      addr = &output[(i*bucketSize)+depthOffset];
      atomicXor(addr, data);
    }
  }
}

#ifdef __cplusplus
}
#endif
